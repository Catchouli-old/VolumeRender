#include "hip/hip_runtime.h"
#include "rendering/Raycast.h"

#ifndef VLR_RAYCAST_CPU

// Define cudacc to get rid of some intellisense errors
#ifndef __HIPCC__
#	define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <hip/device_functions.h>
#include <>
#include <math_functions.h>

#include "maths/Matrix.h"
#include "maths/Normal.h"
#include "maths/Types.h"
#include "resources/Octree.h"
#include "rendering/Rendering.h"
#include "rendering/Raycast.h"
#include "rendering/Shading.h"
#include "util/Util.h"
#include "util/CUDAUtil.h"

#define float_as_int __float_as_int
#define int_as_float __int_as_float

namespace vlr
{
	namespace rendering
	{
		__device__ void raycast(const int32_t* tree, const rendering::ray* ray,
									StackEntry* stack, RaycastHit* raycastHit)
		{
			// The smallest possible positive nonzero floating point number
			const float min_float = exp2f(-MAX_SCALE);

			// Get ray position and direction
			const glm::vec3& origin = ray->origin;
			glm::vec3 dir = ray->direction;

			// Eliminate small (zero) direction values to avoid division by zero
			if (fabsf(dir.x) < min_float)
				dir.x = copysignf(min_float, dir.x);
			if (fabsf(dir.y) < min_float)
				dir.y = copysignf(min_float, dir.y);
			if (fabsf(dir.z) < min_float)
				dir.z = copysignf(min_float, dir.z);

			// Precompute ray equation coefficient and constant for
			// P(t) = o + td
			// Rearranged using inverse expressions of x, y, and z
			float tx_coef = 1.0f / -fabs(dir.x);
			float ty_coef = 1.0f / -fabs(dir.y);
			float tz_coef = 1.0f / -fabs(dir.z);

			float tx_constant = tx_coef * origin.x;
			float ty_constant = ty_coef * origin.y;
			float tz_constant = tz_coef * origin.z;

			// Mirror coordinate system
			// Makes algorithm have fewer cases since ray direction is no longer important
			// + allows for some optimisations later on
			int32_t dir_mask = 7;

			if (dir.x > 0.0f)
			{
				dir_mask ^= (1 << 0);
				tx_constant = 3.0f * tx_coef - tx_constant;
			}

			if (dir.y > 0.0f)
			{
				dir_mask ^= (1 << 1);
				ty_constant = 3.0f * ty_coef - ty_constant;
			}

			if (dir.z > 0.0f)
			{
				dir_mask ^= (1 << 2);
				tz_constant = 3.0f * tz_coef - tz_constant;
			}

			// Compute span of t values for root
			float tx_min = 2.0f * tx_coef - tx_constant;
			float ty_min = 2.0f * ty_coef - ty_constant;
			float tz_min = 2.0f * tz_coef - tz_constant;

			float tx_max = tx_coef - tx_constant;
			float ty_max = ty_coef - ty_constant;
			float tz_max = tz_coef - tz_constant;

			float t_min = fmaxf(fmaxf(tx_min, ty_min), tz_min);
			float t_max = fminf(fminf(tx_max, ty_max), tz_max);

			t_min = fmaxf(t_min, 0.0f);

			float h = t_max;

			// Get root node
			const int32_t* parent = tree + child_desc_size_ints;

			// Evaluate root at centre to get first child node
			int32_t idx = 0;
			float3 pos = { 1.0f, 1.0f, 1.0f };

			int32_t scale = MAX_SCALE - 1;
			float scale_exp2 = 0.5f;

			float tx_centre = 1.5f * tx_coef - tx_constant;
			float ty_centre = 1.5f * ty_coef - ty_constant;
			float tz_centre = 1.5f * tz_coef - tz_constant;

			// Compare centre values against t_min to obtain
			// each bit of idx
			if (tx_centre > t_min)
			{
				idx ^= (1 << 0);
				pos.x = 1.5f;
			}

			if (ty_centre > t_min)
			{
				idx ^= (1 << 1);
				pos.y = 1.5f;
			}

			if (tz_centre > t_min)
			{
				idx ^= (1 << 2);
				pos.z = 1.5f;
			}

			// Cached child descriptor
			int2 child_descriptor = make_int2(0, 0);
			
			// Run until we pop the root voxel
			while (scale < MAX_SCALE)
			{
				// Fetch child descriptor if not valid
				if (child_descriptor.x == 0)
					child_descriptor = *(int2*)parent;

				// Calculate t_max for child
				tx_max = pos.x * tx_coef - tx_constant;
				ty_max = pos.y * ty_coef - ty_constant;
				tz_max = pos.z * tz_coef - tz_constant;

				float t_c_max = fminf(fminf(tx_max, ty_max), tz_max);

				// Mirror idx to get child index
				int32_t child_idx = idx ^ dir_mask;
				int32_t child_mask = child_descriptor.x << child_idx;

				//const raw_attachment* attachment = lookupRawAttachment(tree, parent, 0);
				//glm::vec3 normal = glm::normalize(decompressNormal(attachment->normal));

				// Process voxel if existent and the current span of t values is valid
				if ((child_mask & 0x8000) != 0 && t_min <= t_max)
				// Check the dot product of the normal and the ray direction (basically front face culling)
				//if (glm::dot(glm::vec3(0, 0, -1), normal) > 0.0f)
				//if (attachment != nullptr)
				{
					// TODO:
					// Check if voxel is small enough to terminate traversal
					// (Efficient sparse voxel octrees, Karras and Laine)

					// Find the intersection of t_max and t_c_max
					float tvmax = fminf(t_max, t_c_max);

					// Evaluate child at centre
					float tx_centre = 0.5f * scale_exp2 * tx_coef + tx_max;
					float ty_centre = 0.5f * scale_exp2 * ty_coef + ty_max;
					float tz_centre = 0.5f * scale_exp2 * tz_coef + tz_max;

					// TODO:
					// Implement contours
					// (Efficient sparse voxel octrees, Karras and Laine)

					// Descend if the resulting span is non-zero
					if (t_min <= tvmax)
					{
						// Terminate if this is a leaf voxel
						if ((child_mask & 0x80) == 0)
						{
							break;
						}

						// Write parent voxel and t_max to stack
						if (t_c_max < h)
						{
							stack[scale].parent = parent;
							stack[scale].t_max = t_max;
						}

						// Store h value to eliminate unnecessary stack writes
						h = t_c_max;

						// Update parent voxel
						int32_t ofs = (uint32_t)(child_descriptor.x) >> 17;

						// If this is a far pointer, load it
						if ((child_descriptor.x & 0x10000) != 0)
							ofs = parent[ofs * child_desc_size_ints];

						ofs += get_child_index(child_mask & 0x7F);
						parent += child_desc_size_ints * ofs;

						// Update scale
						scale--;
						scale_exp2 *= 0.5f;

						// Get first child
						idx = 0;

						// Compare t value at centre to get new idx
						if (tx_centre > t_min)
						{
							idx ^= 1;
							pos.x += scale_exp2;
						}

						if (ty_centre > t_min)
						{
							idx ^= 2;
							pos.y += scale_exp2;
						}

						if (tz_centre > t_min)
						{
							idx ^= 4;
							pos.z += scale_exp2;
						}

						// Update max t value
						t_max = tvmax;

						// Invalidate cache child descriptor
						child_descriptor.x = 0;

						continue;
					}
				}

				// Advance the ray
				int32_t step_mask = 0;

				if (tx_max <= t_c_max)
				{
					step_mask ^= (1 << 0);
					pos.x -= scale_exp2;
				}

				if (ty_max <= t_c_max)
				{
					step_mask ^= (1 << 1);
					pos.y -= scale_exp2;
				}
				if (tz_max <= t_c_max)
				{
					step_mask ^= (1 << 2);
					pos.z -= scale_exp2;
				}

				// Update t_min
				t_min = t_c_max;

				// Flip idx
				idx ^= step_mask;

				// Check that direction of flips agree with ray direction
				if ((idx & step_mask) != 0)
				{
					// Pop
					// Find the highest differing bit between pos and oldPos
					uint32_t differing_bits = 0;

					// Opaque bitwise wizardry courtesy of Efficient Sparse Voxel Octrees (Laine and Karras)
					// Get differing bits between each component of pos and oldpos (oldpos.x ^ oldpos.y etc)
					// Then or together the ones which have changed to obtain which bits differ between all three
					if ((step_mask & (1 << 0)) != 0) differing_bits |= __float_as_int(pos.x) ^ __float_as_int(pos.x + scale_exp2);
					if ((step_mask & (1 << 1)) != 0) differing_bits |= __float_as_int(pos.y) ^ __float_as_int(pos.y + scale_exp2);
					if ((step_mask & (1 << 2)) != 0) differing_bits |= __float_as_int(pos.z) ^ __float_as_int(pos.z + scale_exp2);

					// Calculate the scale (the position of the greatest bit)
					scale = (__float_as_int((float)differing_bits) >> 23) - 127;

					// Calculate scale_exp2 (2^(scale - maxScale))
					scale_exp2 = __int_as_float((scale - MAX_SCALE + 127) << 23);

					// Restore parent voxel from the stack.
					StackEntry stackEntry = stack[scale];
					parent = stackEntry.parent;
					t_max = stackEntry.t_max;

					// Get rid of pos values under new scale
					int32_t temp_x = __float_as_int(pos.x) >> scale;
					int32_t temp_y = __float_as_int(pos.y) >> scale;
					int32_t temp_z = __float_as_int(pos.z) >> scale;

					pos.x = __int_as_float(temp_x << scale);
					pos.y = __int_as_float(temp_y << scale);
					pos.z = __int_as_float(temp_z << scale);

					idx = (temp_x & 1) | ((temp_y & 1) << 1) | ((temp_z & 1) << 2);

					// Prevent unnecessary stack writes
					h = 0.0f;
				}
			}

			// Undo mirroring of the coordinate system
			if ((dir_mask & (1 << 0)) == 0) pos.x = 3.0f - scale_exp2 - pos.x;
			if ((dir_mask & (1 << 1)) == 0) pos.y = 3.0f - scale_exp2 - pos.y;
			if ((dir_mask & (1 << 2)) == 0) pos.z = 3.0f - scale_exp2 - pos.z;

			// Output return values
			// Output t of hit
			raycastHit->hit_t = t_min;

			// Output position of hit
			raycastHit->hit_pos.x = fminf(fmaxf(origin.x + t_min * dir.x, pos.x + min_float), pos.x + scale_exp2 - min_float);
			raycastHit->hit_pos.y = fminf(fmaxf(origin.y + t_min * dir.y, pos.y + min_float), pos.y + scale_exp2 - min_float);
			raycastHit->hit_pos.z = fminf(fmaxf(origin.z + t_min * dir.z, pos.z + min_float), pos.z + scale_exp2 - min_float);

			// Output parent of hit voxel
			raycastHit->hit_parent = parent;

			// Output child index of hit voxel
			raycastHit->hit_idx = idx ^ (dir_mask ^ 7);
			raycastHit->hit_pos_internal = *(glm::vec3*)&pos;

			// Output scale of hit voxel
			raycastHit->hit_scale = scale;
		}

		// TODO: restore old state
		__device__ void raycast_empty(const int32_t* tree, const rendering::ray* ray, StackEntry* stack,
								RaycastHit* raycastHit, const RaycastHit* old_hit)
		{
			const float MAX_REFRACTION_STEP = 100.0f;

			// The smallest possible positive nonzero floating point number
			const float min_float = exp2f(-MAX_SCALE);

			// Get ray position and direction
			const glm::vec3& origin = ray->origin;
			glm::vec3 dir = ray->direction;

			// Eliminate small (zero) direction values to avoid division by zero
			if (fabsf(dir.x) < min_float)
				dir.x = copysignf(min_float, dir.x);
			if (fabsf(dir.y) < min_float)
				dir.y = copysignf(min_float, dir.y);
			if (fabsf(dir.z) < min_float)
				dir.z = copysignf(min_float, dir.z);

			// Precompute ray equation coefficient and constant for
			// P(t) = o + td
			// Rearranged using inverse expressions of x, y, and z
			float tx_coef = 1.0f / -fabs(dir.x);
			float ty_coef = 1.0f / -fabs(dir.y);
			float tz_coef = 1.0f / -fabs(dir.z);

			float tx_constant = tx_coef * origin.x;
			float ty_constant = ty_coef * origin.y;
			float tz_constant = tz_coef * origin.z;

			// Mirror coordinate system
			// Makes algorithm have fewer cases since ray direction is no longer important
			// + allows for some optimisations later on
			int32_t dir_mask = 7;

			if (dir.x > 0.0f)
			{
				dir_mask ^= (1 << 0);
				tx_constant = 3.0f * tx_coef - tx_constant;
			}

			if (dir.y > 0.0f)
			{
				dir_mask ^= (1 << 1);
				ty_constant = 3.0f * ty_coef - ty_constant;
			}

			if (dir.z > 0.0f)
			{
				dir_mask ^= (1 << 2);
				tz_constant = 3.0f * tz_coef - tz_constant;
			}

			// Compute span of t values for root
			float tx_min = 2.0f * tx_coef - tx_constant;
			float ty_min = 2.0f * ty_coef - ty_constant;
			float tz_min = 2.0f * tz_coef - tz_constant;

			float tx_max = tx_coef - tx_constant;
			float ty_max = ty_coef - ty_constant;
			float tz_max = tz_coef - tz_constant;

			float t_min = fmaxf(fmaxf(tx_min, ty_min), tz_min);
			float t_max = fminf(fminf(tx_max, ty_max), tz_max);

			t_min = fmaxf(t_min, 0.0f);
			t_max = fminf(t_max, t_min + MAX_REFRACTION_STEP);

			float h = t_max;

			// Get root node
			const int32_t* parent = tree + child_desc_size_ints;

			// Evaluate root at centre to get first child node
			int32_t idx = 0;
			float3 pos = { 1.0f, 1.0f, 1.0f };

			// Store old parent & idx in case we need to return them
			const int32_t* old_parent = parent;
			int32_t old_idx = idx;
			float3 old_pos = pos;

			int32_t scale = MAX_SCALE - 1;
			float scale_exp2 = 0.5f;

			float tx_centre = 1.5f * tx_coef - tx_constant;
			float ty_centre = 1.5f * ty_coef - ty_constant;
			float tz_centre = 1.5f * tz_coef - tz_constant;

			// Compare centre values against t_min to obtain
			// each bit of idx
			if (tx_centre > t_min)
			{
				idx ^= (1 << 0);
				pos.x = 1.5f;
			}

			if (ty_centre > t_min)
			{
				idx ^= (1 << 1);
				pos.y = 1.5f;
			}

			if (tz_centre > t_min)
			{
				idx ^= (1 << 2);
				pos.z = 1.5f;
			}

			// Cached child descriptor
			int2 child_descriptor = make_int2(0, 0);
			
			// Run until we pop the root voxel
			while (scale < MAX_SCALE)
			{
				// Fetch child descriptor if not valid
				if (child_descriptor.x == 0)
					child_descriptor = *(int2*)parent;

				// Calculate t_max for child
				tx_max = pos.x * tx_coef - tx_constant;
				ty_max = pos.y * ty_coef - ty_constant;
				tz_max = pos.z * tz_coef - tz_constant;

				float t_c_max = fminf(fminf(tx_max, ty_max), tz_max);

				// Mirror idx to get child index
				int32_t child_idx = idx ^ dir_mask;
				int32_t child_mask = child_descriptor.x << child_idx;
				//const raw_attachment* attachment = lookupRawAttachment(tree, parent, 0);
				//glm::vec3 normal = glm::normalize(decompressNormal(attachment->normal));

				if ((child_mask & 0x8000) != 0 && t_min <= t_max)
				{
					// Store old parent & idx in case we need to return them
					old_parent = parent;
					old_idx = idx;
					old_pos = pos;

					// TODO: contours (laine and karras)
					// If this voxel has a contour
					int contour_mask = child_descriptor.y;
					if ((contour_mask & 0x80) != 0)
					{

					}

					// Find the intersection of t_max and t_c_max
					float tvmax = fminf(t_max, t_c_max);

					// Evaluate child at centre
					float tx_centre = 0.5f * scale_exp2 * tx_coef + tx_max;
					float ty_centre = 0.5f * scale_exp2 * ty_coef + ty_max;
					float tz_centre = 0.5f * scale_exp2 * tz_coef + tz_max;

					// TODO:
					// Implement contours
					// (Efficient sparse voxel octrees, Karras and Laine)

					// Descend if the resulting span is non-zero
					if (t_min <= tvmax)
					{
						// Push if this is not a leaf voxel
						if ((child_mask & 0x80) != 0)
						{
							// Write parent voxel and t_max to stack
							if (t_c_max < h)
							{
								stack[scale].parent = parent;
								stack[scale].t_max = t_max;
							}

							// Store h value to eliminate unnecessary stack writes
							h = t_c_max;

							// Update parent voxel
							int32_t ofs = (uint32_t)(child_descriptor.x) >> 17;

							// If this is a far pointer, load it
							if ((child_descriptor.x & 0x10000) != 0)
								ofs = parent[ofs * child_desc_size_ints];

							ofs += get_child_index(child_mask & 0x7F);
							parent += child_desc_size_ints * ofs;

							// Update scale
							scale--;
							scale_exp2 *= 0.5f;

							// Get first child
							idx = 0;

							// Compare t value at centre to get new idx
							if (tx_centre > t_min)
							{
								idx ^= 1;
								pos.x += scale_exp2;
							}

							if (ty_centre > t_min)
							{
								idx ^= 2;
								pos.y += scale_exp2;
							}

							if (tz_centre > t_min)
							{
								idx ^= 4;
								pos.z += scale_exp2;
							}

							// Update max t value
							t_max = tvmax;

							// Invalidate cache child descriptor
							child_descriptor.x = 0;

							continue;
						}
					}
				}

				// Advance the ray
				int32_t step_mask = 0;

				if (tx_max <= t_c_max)
				{
					step_mask ^= (1 << 0);
					pos.x -= scale_exp2;
				}

				if (ty_max <= t_c_max)
				{
					step_mask ^= (1 << 1);
					pos.y -= scale_exp2;
				}
				if (tz_max <= t_c_max)
				{
					step_mask ^= (1 << 2);
					pos.z -= scale_exp2;
				}

				// Update t_min
				t_min = t_c_max;

				// Flip idx
				idx ^= step_mask;

				//printf("%f\n", t_c_max);

				// Check that direction of flips agree with ray direction
				if ((idx & step_mask) != 0)
				{
					// Pop
					// Find the highest differing bit between pos and oldPos
					uint32_t differing_bits = 0;

					// Opaque bitwise wizardry courtesy of Efficient Sparse Voxel Octrees (Laine and Karras)
					// Get differing bits between each component of pos and oldpos (oldpos.x ^ oldpos.y etc)
					// Then or together the ones which have changed to obtain which bits differ between all three
					if ((step_mask & (1 << 0)) != 0) differing_bits |= float_as_int(pos.x) ^ float_as_int(pos.x + scale_exp2);
					if ((step_mask & (1 << 1)) != 0) differing_bits |= float_as_int(pos.y) ^ float_as_int(pos.y + scale_exp2);
					if ((step_mask & (1 << 2)) != 0) differing_bits |= float_as_int(pos.z) ^ float_as_int(pos.z + scale_exp2);

					// Calculate the scale (the position of the greatest bit)
					scale = (float_as_int((float)differing_bits) >> 23) - 127;

					// Calculate scale_exp2 (2^(scale - maxScale))
					scale_exp2 = int_as_float((scale - MAX_SCALE + 127) << 23);

					// Restore parent voxel from the stack.
					StackEntry stackEntry = stack[scale];
					parent = stackEntry.parent;
					t_max = stackEntry.t_max;

					// Get rid of pos values under new scale
					int32_t temp_x = float_as_int(pos.x) >> scale;
					int32_t temp_y = float_as_int(pos.y) >> scale;
					int32_t temp_z = float_as_int(pos.z) >> scale;

					pos.x = int_as_float(temp_x << scale);
					pos.y = int_as_float(temp_y << scale);
					pos.z = int_as_float(temp_z << scale);

					// Compute new idx
					idx = (temp_x & 1) | ((temp_y & 1) << 1) | ((temp_z & 1) << 2);

					// Invalidate cached descriptor
					child_descriptor.x = 0;

					// Prevent unnecessary stack writes
					h = 0.0f;
				}

				if (scale != 23)
				{
					// Read child desc
					int32_t cdesc = *(int32_t*)parent;

					// If this voxel does not exist
					if (((cdesc << (idx ^ dir_mask)) & 0x8000) == 0)
					{
						// Restore last existent voxel and return
						idx = old_idx;
						pos = old_pos;
						parent = old_parent;

						break;
					}
				}
			}

			// Undo mirroring of the coordinate system
			if ((dir_mask & (1 << 0)) == 0) pos.x = 3.0f - scale_exp2 - pos.x;
			if ((dir_mask & (1 << 1)) == 0) pos.y = 3.0f - scale_exp2 - pos.y;
			if ((dir_mask & (1 << 2)) == 0) pos.z = 3.0f - scale_exp2 - pos.z;

			// Output return values
			// Output t of hit
			raycastHit->hit_t = t_min;

			// Output position of hit
			raycastHit->hit_pos.x = fminf(fmaxf(origin.x + t_min * dir.x, pos.x + min_float), pos.x + scale_exp2 - min_float);
			raycastHit->hit_pos.y = fminf(fmaxf(origin.y + t_min * dir.y, pos.y + min_float), pos.y + scale_exp2 - min_float);
			raycastHit->hit_pos.z = fminf(fmaxf(origin.z + t_min * dir.z, pos.z + min_float), pos.z + scale_exp2 - min_float);

			// Output parent of hit voxel
			raycastHit->hit_parent = parent;

			// Output child index of hit voxel
			raycastHit->hit_idx = idx ^ (dir_mask ^ 7);
			raycastHit->hit_pos_internal = *(glm::vec3*)&pos;

			// Output scale of hit voxel
			raycastHit->hit_scale = scale;
		}
	}
}

#endif /* VLR_RAYCAST_CPU */
